#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <>
#include <vector>
#include <cstdlib>
#include <ctime>

// Constants
const int NUM_PARTICLES = 1023;
const float RADIUS = 0.01f;
const int GRID_SIZE = 64;
const float WORLD_SIZE = 2.0f; // from -1 to 1

struct Particle {
    float x, y;
    float vx, vy;
};

// CUDA hash function for grid cell index
__device__ int getCellIndex(float x, float y) {
    int cx = (int)((x + 1.0f) / WORLD_SIZE * GRID_SIZE);
    int cy = (int)((y + 1.0f) / WORLD_SIZE * GRID_SIZE);
    cx = max(0, min(GRID_SIZE - 1, cx));
    cy = max(0, min(GRID_SIZE - 1, cy));
    return cy * GRID_SIZE + cx;
}

// Kernel to reset grid counts to zero
__global__ void resetGridCounts(int* gridCount) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < GRID_SIZE * GRID_SIZE) {
        gridCount[idx] = 0;
    }
}

// Kernel to assign particles to grid cells and update positions
__global__ void simulateParticles(Particle* particles, int* grid, int* gridCount) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= NUM_PARTICLES) return;

    Particle& p = particles[i];

    // Update position
    p.x += p.vx;
    p.y += p.vy;

    // Bounce off walls
    if (p.x > 1.0f || p.x < -1.0f) p.vx = -p.vx;
    if (p.y > 1.0f || p.y < -1.0f) p.vy = -p.vy;

    // Compute grid cell
    int cell = getCellIndex(p.x, p.y);

    // Insert particle index atomically into grid cell
    int index = atomicAdd(&gridCount[cell], 1);
    if (index < NUM_PARTICLES) {  // avoid overflow
        grid[cell * NUM_PARTICLES + index] = i;
    }
}

// Kernel to handle collisions per particle
__global__ void collideParticles(Particle* particles, int* grid, int* gridCount) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= NUM_PARTICLES) return;

    Particle& p = particles[i];
    int cell = getCellIndex(p.x, p.y);
    int count = gridCount[cell];

    for (int j = 0; j < count; ++j) {
        int otherIdx = grid[cell * NUM_PARTICLES + j];
        if (otherIdx <= i) continue; // prevent double processing and self

        Particle& q = particles[otherIdx];

        float dx = p.x - q.x;
        float dy = p.y - q.y;
        float distSq = dx * dx + dy * dy;
        float minDist = 2.0f * RADIUS;

        if (distSq < minDist * minDist) {
            float dist = sqrtf(distSq);
            if (dist == 0.0f) continue; // avoid division by zero

            float nx = dx / dist;
            float ny = dy / dist;

            // Relative velocity
            float dvx = p.vx - q.vx;
            float dvy = p.vy - q.vy;

            // Project relative velocity onto collision normal
            float relVel = dvx * nx + dvy * ny;

            if (relVel < 0.0f) {
                // Apply elastic response (equal mass case)
                float impulse = relVel;

                p.vx -= impulse * nx;
                p.vy -= impulse * ny;
                q.vx += impulse * nx;
                q.vy += impulse * ny;

                // Optional: separate overlapping particles to avoid sticking
                float overlap = 0.5f * (minDist - dist);
                p.x += nx * overlap;
                p.y += ny * overlap;
                q.x -= nx * overlap;
                q.y -= ny * overlap;
            }
        }
    }
}

// Host-side data
std::vector<Particle> h_particles;

// Device pointers
Particle* d_particles = nullptr;
int* d_grid = nullptr;
int* d_gridCount = nullptr;

void initParticles() {
    h_particles.resize(NUM_PARTICLES);
    for (auto& p : h_particles) {
        p.x = (rand() % 2000 - 1000) / 1000.0f;
        p.y = (rand() % 2000 - 1000) / 1000.0f;
        p.vx = ((rand() % 2000) - 1000) / 100000.0f;
        p.vy = ((rand() % 2000) - 1000) / 100000.0f;
    }

    hipMemcpy(d_particles, h_particles.data(), sizeof(Particle) * NUM_PARTICLES, hipMemcpyHostToDevice);
}

void updateParticlesCUDA() {
    // Reset grid counts
    resetGridCounts<<<(GRID_SIZE * GRID_SIZE + 255) / 256, 256>>>(d_gridCount);

    // Simulate motion & assign particles to grid cells
    simulateParticles<<<(NUM_PARTICLES + 255) / 256, 256>>>(d_particles, d_grid, d_gridCount);

    // Wait for motion update and grid assignment
    hipDeviceSynchronize();

    // Handle collisions per particle
    collideParticles<<<(NUM_PARTICLES + 255) / 256, 256>>>(d_particles, d_grid, d_gridCount);

    // Copy back to host
    hipMemcpy(h_particles.data(), d_particles, sizeof(Particle) * NUM_PARTICLES, hipMemcpyDeviceToHost);
}

void drawParticles() {
    glPointSize(10.0f);
    glBegin(GL_POINTS);
    glColor3f(1.0f, 0.0f, 0.0f);
    for (const auto& p : h_particles) {
        glVertex2f(p.x, p.y);
    }
    glEnd();
}

int main() {
    srand((unsigned)time(0));

    if (!glfwInit()) return -1;
    GLFWwindow* window = glfwCreateWindow(800, 600, "CUDA Ideal Gas", NULL, NULL);
    if (!window) { glfwTerminate(); return -1; }
    glfwMakeContextCurrent(window);
    glewInit();

    // Allocate device memory
    hipMalloc(&d_particles, sizeof(Particle) * NUM_PARTICLES);
    hipMalloc(&d_grid, sizeof(int) * GRID_SIZE * GRID_SIZE * NUM_PARTICLES);
    hipMalloc(&d_gridCount, sizeof(int) * GRID_SIZE * GRID_SIZE);

    initParticles();

    while (!glfwWindowShouldClose(window)) {
        int width, height;
        glfwGetFramebufferSize(window, &width, &height);
        glViewport(0, 0, width, height);

        updateParticlesCUDA();

        glClearColor(0, 0, 0, 1);
        glClear(GL_COLOR_BUFFER_BIT);
        drawParticles();

        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    // Cleanup
    hipFree(d_particles);
    hipFree(d_grid);
    hipFree(d_gridCount);

    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}

